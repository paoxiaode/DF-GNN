#include "../sddmm/sddmm.cuh"
#include "../spmm/spmm.cuh"
#include "../util/computeUtil.h"
#include "../util/hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <torch/types.h>
#include <unistd.h>

void gt_softmax_inference_launch(int m, int nnz, int h, int f, int smem_consume,
                                 const int *indptr, const int *indices,
                                 const int *rows, const float *val,
                                 const float *Q, const float *K, const float *V,
                                 float *attn_edge, float *out_feat) {
  const int ntx = 32; // on feature dimension
  const int nty = 8;  // on out dimension
  const int nbx = (nnz + nty - 1) / nty;
  const int nby = FindNumBlocks<'y'>(h);
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);

  CUDA_KERNEL_CALL((sddmmCooKernel<float>), nblks, nthrs, 0, f * h, f * h, h,
                   nnz, f, rows, indices, val, Q, K, attn_edge);

  const dim3 nblks2(m, h, 1);
  const dim3 nthrs2(32, (f + 31) / 32, 1);
  //   CUDA_KERNEL_CALL((sddmmCsrKernel<float>), nblks2, nthrs2,
  //                    0, h, f, indptr, indices, val, Q, K, attn_edge);
  CUDA_KERNEL_CALL((softMax_SPMM<float>), nblks2, nthrs2,
                   (smem_consume) * sizeof(float), h, f, indptr, indices, V,
                   attn_edge, out_feat);
}

std::vector<torch::Tensor>
gt_softmax_inference_cuda(torch::Tensor indptr, torch::Tensor indices,
                          torch::Tensor rows, torch::Tensor val,
                          int smem_consume, torch::Tensor Q, torch::Tensor K,
                          torch::Tensor V) {
  const auto m = indptr.size(0) - 1; // num of nodes
  const auto nnz = indices.size(0);  // num of edges
  const auto h = Q.size(1);          // num of heads
  const auto f = Q.size(2);          // num of feats
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  auto attn_edge = torch::zeros({nnz * h}, options);
  gt_softmax_inference_launch(
      m, nnz, h, f, smem_consume, indptr.data_ptr<int>(),
      indices.data_ptr<int>(), rows.data_ptr<int>(), val.data_ptr<float>(),
      Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
      attn_edge.data_ptr<float>(), out_feat.data_ptr<float>());
  return {out_feat};
}