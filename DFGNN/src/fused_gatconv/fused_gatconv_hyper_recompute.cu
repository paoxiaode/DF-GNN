#include "hip/hip_runtime.h"
#include "../util/computeUtil.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>

// template <typename DType>
// __global__ void fused_gat_hyper_recompute_inference(
//     int m, int h, int f, const DType *attn_row, const DType *attn_col,
//     const int *row, const int *indptr, const int *indices, const DType
//     *in_feat, const DType negative_slope, DType *out_feat) {
//   // launch dim (32, 8) * (num_nodes/8, 1)
//   const int bidx = blockIdx.x;
//   const int hid = blockIdx.y;
//   const int tidx = threadIdx.x;
//   const int tidy = threadIdx.y;
//   const int tid = tidy * 32 + tidx;

//   // the node bound of this block
//   const int blockSize = blockDim.y;
//   const int blk_node_lb = blockSize * bidx;
//   const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

//   // the edge bound of this block
//   const int blk_edge_lb = indptr[blk_node_lb];
//   const int blk_edge_hb = indptr[blk_node_hb];

//   // the num of edges in this block
//   const int blk_num_edge = blk_edge_hb - blk_edge_lb;

//   // init smem
//   extern __shared__ DType smem[];
//   DType *neigh_nodes_weight = smem; // [8, f]

//   const int *rowoff = row + blk_edge_lb;
//   const int *indicesoff = indices + blk_edge_lb;

//   // SDDMM, edge parallel
//   for (int i = tid; i < blk_num_edge; i += blockSize * WARP_SIZE) {
//     if (i < blk_num_edge) {
//       const int src = __ldg(rowoff + i);
//       const int dst = __ldg(indicesoff + i);
//       DType weight = attn_row[src * h + hid] + attn_col[dst * h + hid];
//       weight = LeakyRelu(weight, negative_slope);
//       neigh_nodes_weight[i] = weight;
//     }
//   }
//   __syncthreads();

//   // Softmax+SPMM, node parallel
//   int curr_node = blk_node_lb + tidy;
//   if (curr_node < blk_node_hb) {
//     const int edge_lb = indptr[curr_node];
//     const int edge_hb = indptr[curr_node + 1];
//     const int num_edge = edge_hb - edge_lb;

//     DType weightMax = -1e38;
//     const int hf = h * f;
//     // const int hfid = hid * f + tidx;

//     DType *neigh_nodes_weight_off =
//         neigh_nodes_weight + (edge_lb - blk_edge_lb);

//     int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
//     for (int j = 0; j < loop; j++) {
//       DType weight = -1e38;
//       int pid = tidx + (j << 5);
//       if (pid < num_edge) {
//         weight = neigh_nodes_weight_off[pid];
//       }
//       __syncwarp();
// #pragma unroll
//       for (int stride = 16; stride > 0; stride >>= 1) {
//         weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32),
//         weight);
//       }
//       __syncwarp();
//       weightMax = MAX(weight, weightMax);
//     }

//     // compute the sum of exp
//     DType expAll = 0;
//     for (int j = 0; j < loop; j++) {
//       int pid = tidx + (j << 5); // node need to process in loop j
//       DType exptmp = 0;
//       if (pid < num_edge) {
//         DType weight = neigh_nodes_weight_off[pid];
//         exptmp = exp(weight - weightMax);
//         neigh_nodes_weight_off[pid] = exptmp;
//       }
//       __syncwarp();
// #pragma unroll
//       for (int stride = 16; stride > 0; stride >>= 1) {
//         exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
//       }
//       __syncwarp();
//       expAll += exptmp;
//     }
//     expAll = (expAll != 0) ? 1.0f / expAll : 0;

//     // compute the output
//     int loop_f = (f + WARP_SIZE - 1) / WARP_SIZE;
//     for (int i = 0; i < loop_f; i++) {
//       DType acc = 0;
//       int pid = tidx + (i << 5);
//       for (int j = 0; j < num_edge; j++) {
//         int cid = indices[edge_lb + j];
//         DType attn_val = neigh_nodes_weight_off[j];
//         if (pid < f)
//           acc += attn_val * in_feat[cid * hf + hid * f + pid];
//       }
//       // handle the node with no neighbor
//       if (pid < f)
//         out_feat[curr_node * hf + hid * f + pid] = acc * expAll;
//     }
//   }
// }

template <typename DType>
__global__ void fused_gat_hyper_recompute_inference_vec4(
    int m, int h, int f, const DType *attn_row, const DType *attn_col,
    const int *indptr, const int *indices, const DType *in_feat,
    const DType negative_slope, DType *out_feat) {
  // launch dim (32, 8) * (num_nodes/8, 1)
  const int bidx = blockIdx.x;
  const int hid = blockIdx.y;
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int tid = tidy * 32 + tidx;

  // the node bound of this block
  const int blockSize = blockDim.y;
  const int blk_node_lb = blockSize * bidx;
  const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

  // the edge bound of this block
  const int blk_edge_lb = indptr[blk_node_lb];

  // Softmax+SPMM, node parallel
  const int curr_node = blk_node_lb + tidy;
  if (curr_node < blk_node_hb) {
    const int edge_lb = indptr[curr_node];
    const int edge_hb = indptr[curr_node + 1];
    const int num_edge = edge_hb - edge_lb;
    const int *indicesoff = indices + edge_lb;

    DType attn_row_val = attn_row[curr_node * h + hid];
    DType weightMax = -1e38;
    const int hf = h * f;
    int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
    for (int j = 0; j < loop; j++) {
      DType weight = -1e38;
      int pid = tidx + (j << 5);
      if (pid < num_edge) {
        int cid = __ldg(indicesoff + pid);
        DType attn_col_val = attn_col[cid * h + hid];
        weight = attn_row_val + attn_col_val;
        weight = LeakyRelu(weight, negative_slope);
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32), weight);
      }
      __syncwarp();
      weightMax = MAX(weight, weightMax);
    }

    // compute the sum of exp
    DType expAll = 0;
    for (int j = 0; j < loop; j++) {
      int pid = tidx + (j << 5); // node need to process in loop j
      DType exptmp = 0;
      if (pid < num_edge) {
        int cid = __ldg(indicesoff + pid);
        DType attn_col_val = attn_col[cid * h + hid];
        DType weight = attn_row_val + attn_col_val;
        weight = LeakyRelu(weight, negative_slope);
        exptmp = exp(weight - weightMax);
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }
    expAll = (expAll != 0) ? 1.0f / expAll : 0;

    DType *Outoff = out_feat + curr_node * h * f + hid * f;
    for (int i = tidx; i < f / 4; i += 32) {
      DType weight = 0;
      DType acc[4] = {0, 0, 0, 0};
      for (int j = 0; j < num_edge; j++) {
        int cid = __ldg(indicesoff + j);
        DType attn_col_val = attn_col[cid * h + hid];
        weight = attn_row_val + attn_col_val;
        weight = LeakyRelu(weight, negative_slope);
        weight = exp(weight - weightMax);
        const DType *featoff = in_feat + cid * h * f + hid * f + 4 * i;
        Mul4_const<float>(acc, featoff, weight);
      }
      // handle the node with no neighbor
      selfMulConst4<float>(acc, expAll);
      Store<float4, float>(Outoff, acc, 4 * i);
    }
  }
}

void gat_hyper_recompute_inference_launch(
    int m, int nnz, int h, int f, const float *attn_row, const float *attn_col,
    const int *indptr, const int *indices, float negative_slope,
    const float *in_feat, float *out_feat) {
  const int ntx = 32;
  const int nty = 8;

  const int nbx = (m + nty - 1) / nty;
  const int nby = h;
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);

  if ((f % 128) == 0) {
    CUDA_KERNEL_CALL((fused_gat_hyper_recompute_inference_vec4<float>), nblks,
                     nthrs, 0, m, h, f, attn_row, attn_col, indptr, indices,
                     in_feat, negative_slope, out_feat);
  } else {
    // TODO: not implemented
    exit(0);
    // CUDA_KERNEL_CALL((fused_gat_hyper_recompute_inference<float>), nblks,
    // nthrs,
    //                  smem_size, m, h, f, attn_row, attn_col, indptr,
    //                  indices, in_feat, negative_slope, out_feat);
  }
}

torch::Tensor gat_hyper_recompute_inference_cuda(
    torch::Tensor attn_row, torch::Tensor attn_col, torch::Tensor indptr,
    torch::Tensor indices, float negative_slope, torch::Tensor in_feat) {
  const auto m = indptr.size(0) - 1;
  const auto nnz = indices.size(0);
  const auto h = attn_row.size(1);
  const auto f = in_feat.size(2);
  auto devid = attn_row.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::empty({m, h, f}, options);
  gat_hyper_recompute_inference_launch(
      m, nnz, h, f, attn_row.data_ptr<float>(), attn_col.data_ptr<float>(),
      indptr.data_ptr<int>(), indices.data_ptr<int>(), negative_slope,
      in_feat.data_ptr<float>(), out_feat.data_ptr<float>());
  return out_feat;
}