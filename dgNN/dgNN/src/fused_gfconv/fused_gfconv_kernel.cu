#include "hip/hip_runtime.h"
#include "../util/computeUtil.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>

#include <unistd.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define CURAND_CALL(x)                                \
  do                                                  \
  {                                                   \
    if ((x) != HIPRAND_STATUS_SUCCESS)                 \
    {                                                 \
      printf("Error at %s:%d\n", __FILE__, __LINE__); \
      return EXIT_FAILURE;                            \
    }                                                 \
  } while (0)

#define CUDA_KERNEL_CALL(kernel, nblks, nthrs, shmem, ...)          \
  {                                                                 \
    {                                                               \
      (kernel)<<<(nblks), (nthrs), (shmem)>>>(__VA_ARGS__);         \
      hipError_t e = hipGetLastError();                           \
      CHECK(e == hipSuccess || e == hipErrorDeinitialized)      \
          << "CUDA kernel launch error: " << hipGetErrorString(e); \
    }                                                               \
  }

__global__ void fused_forward_kernel(const int m, const int nnz, const int h, const int f,
                                     const int *row_ptr, const int *col_ind, const float *val,
                                     const float *Q, const float *K, const float *V,
                                     float *out_feat)
{
  int rid = blockIdx.x;                     // loop over row of adj matrix
  int hid = blockIdx.y;                     // loop over heads
  int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  int lb = row_ptr[rid]; // row rid elements
  int hb = row_ptr[rid + 1];
  int ptr = threadIdx.x; // the neighbor node needed to process

  int threads_x = blockDim.x; // 32
  int threads_y = blockDim.y; // f/32
  int blockSize = threads_x * threads_y;
  int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *curr_node_feature = smem;
  float *feat_prod_result = (float *)&curr_node_feature[f];
  float *neigh_nodes_weight = (float *)&feat_prod_result[f];
  float weightMax = -1e38;

  // init the shared memory
  if (fid < f)
  {
    curr_node_feature[fid] = Q[rid * h * f + hid * f + fid];
  }

  // compute the attention weight
  for (int j = 0; j < num_neighbor; j++)
  {
    float weight;
    float weight_partial = 0;
    if (fid < f)
    {
      int cid = col_ind[lb + j];
      weight_partial = curr_node_feature[fid] * K[cid * h * f + hid * f + fid];
      feat_prod_result[fid] = weight_partial;
    }
    __syncthreads();
    if (fid < 32)
    {
      volatile float *sdata = feat_prod_result;
      if (blockSize >= 64)
        sdata[fid] += sdata[fid + 32];
      if (blockSize >= 32)
        sdata[fid] += sdata[fid + 16];
      if (blockSize >= 16)
        sdata[fid] += sdata[fid + 8];
      if (blockSize >= 8)
        sdata[fid] += sdata[fid + 4];
      if (blockSize >= 4)
        sdata[fid] += sdata[fid + 2];
      if (blockSize >= 2)
        sdata[fid] += sdata[fid + 1];
      __syncwarp();
      if (fid == 0)
      {
        neigh_nodes_weight[j] = sdata[0];
      }
      __syncwarp();
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = ptr + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < hb - lb)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float weight;
    float attn_val;
    int cid = col_ind[lb + j];
    if (fid < f)
    {
      float weight = neigh_nodes_weight[j];
      attn_val = exp(weight - weightMax) / expAll;
      acc += attn_val * V[cid * h * f + hid * f + fid];
    }
    __syncthreads();
  }
  if (fid < f)
    out_feat[rid * h * f + hid * f + fid] = acc;
}

void gf_forward(int m, int nnz, int h, int f,
                const int *row_ptr, const int *col_ind, const float *val,
                const float *Q, const float *K, const float *V,
                float *out_feat)
{
  // float rt;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  const dim3 nblks(m, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  // printf("start kernel\n");
  // CUDA_KERNEL_CALL(
  //     (fused_forward_kernel),
  //     nblks, nthrs, (f + m) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
  //     Q, K, V, edge_max, edge_sum, edge_mask, out_feat, seed);
  fused_forward_kernel<<<dim3(m, h, 1), dim3(32, (f + 31) / 32, 1),
                         (2 * f + 100) * sizeof(float)>>>(
      m, nnz, h, f, row_ptr, col_ind, val,
      Q, K, V, out_feat);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time of fused kernel: %f \n", elapsedTime);

  hipError_t errSync = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

std::vector<torch::Tensor>
gf_forward_cuda(torch::Tensor row_ptr,
                torch::Tensor col_ind,
                torch::Tensor val, torch::Tensor Q,
                torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = row_ptr.size(0) - 1; // num nodes
  const auto nnz = col_ind.size(0);   // num edges
  const auto h = Q.size(1);           // num heads
  const auto f = Q.size(2);           // num feats
  auto devid = row_ptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  gf_forward(m, nnz, h, f,
             row_ptr.data_ptr<int>(), col_ind.data_ptr<int>(), val.data_ptr<float>(),
             Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
             out_feat.data_ptr<float>());
  return {out_feat};
}