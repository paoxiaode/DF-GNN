#include "hip/hip_runtime.h"
#include "../util/computeUtil.h"
#include "../util/hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <torch/types.h>
#include <unistd.h>

using namespace std;

template <typename DType>
__global__ void fused_gt_hyper(const int m, const int nnz, const int h,
                               const int f, const int *row, const int *indptr,
                               const int *indices, const DType *val,
                               const DType *Q, const DType *K, const DType *V,
                               DType *attn_edge, DType *out_feat) {
  // launch dim (32, 8) * (num_nodes/8, 1)

  const int bidx = blockIdx.x;
  const int hid = blockIdx.y;
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;

  // the node bound of this block
  const int blockSize = blockDim.y;
  const int blk_node_lb = blockSize * bidx;
  const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

  // the edge bound of this block
  const int blk_edge_lb = indptr[blk_node_lb];
  const int blk_edge_hb = indptr[blk_node_hb];

  // the num of edges in this block
  const int blk_num_edge = blk_edge_hb - blk_edge_lb;

  // init smem
  extern __shared__ DType smem[];
  DType *neigh_nodes_weight = smem; // [8, f]

  // SDDMM, edge parallel
  int nnz_per_warp = (blk_num_edge + blockSize - 1) / blockSize;

  const int *rowoff = row + blk_edge_lb;
  const int *indicesoff = indices + blk_edge_lb;
  const DType *valoff = val + blk_edge_lb;

  int src;
  int dst;
  for (int i = 0; i < nnz_per_warp; i++) {
    int curr_edge = tidy * nnz_per_warp + i;
    // edge bound for curr block
    if (curr_edge < blk_num_edge) {
      src = __ldg(rowoff + curr_edge);
      dst = __ldg(indicesoff + curr_edge);

      // // the Q feature of row node
      const DType *Qoff = Q + src * f * h + hid * f;
      // the K feature of col node
      const DType *Koff = K + dst * f * h + hid * f;

      DType att_val = 0;
      for (int j = tidx; j < f; j += 64) {
        // float2 Q2 = reinterpret_cast<const float2*>(Qoff)[j];
        // float2 K2 = reinterpret_cast<const float2*>(Koff)[j];
        // att_val += vecDot2<float2, float>(Q2, K2);
        att_val += Qoff[j] * Koff[j];
        if (j + 32 < f)
          att_val += Qoff[j + 32] * Koff[j + 32];
      }
#pragma unroll
      for (int offset = 16; offset > 0; offset /= 2)
        att_val += __shfl_down_sync(full_mask, att_val, offset);
      if (tidx == 0) {
        // TODO consider to move val into smem
        neigh_nodes_weight[curr_edge] = att_val * valoff[curr_edge];
      }
    }
  }
  __syncthreads();

  // Softmax+SPMM, node parallel
  int curr_node = blk_node_lb + tidy;
  if (curr_node < blk_node_hb) {
    const int edge_lb = indptr[curr_node];
    const int edge_hb = indptr[curr_node + 1];
    const int num_edge = edge_hb - edge_lb;

    DType weightMax = -1e38;
    const int hf = h * f;
    // const int hfid = hid * f + tidx;

    DType *neigh_nodes_weight_off =
        neigh_nodes_weight + (edge_lb - blk_edge_lb);

    int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
    for (int j = 0; j < loop; j++) {
      DType weight = -1e38;
      int pid = tidx + (j << 5);
      if (pid < num_edge) {
        weight = neigh_nodes_weight_off[pid];
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32), weight);
      }
      __syncwarp();
      weightMax = MAX(weight, weightMax);
    }
    // compute the sum of exp
    DType expAll = 0;
    for (int j = 0; j < loop; j++) {
      int pid = tidx + (j << 5); // node need to process in loop j
      DType exptmp = 0;
      if (pid < num_edge) {
        DType weight = neigh_nodes_weight_off[pid];
        exptmp = exp(weight - weightMax);
        neigh_nodes_weight_off[pid] = exptmp;
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }

    // compute the output
    int loop_f = (f + WARP_SIZE - 1) / WARP_SIZE;
    for (int i = 0; i < loop_f; i += 1) {
      DType acc = 0;
      int pid = tidx + (i << 5);
      for (int j = 0; j < num_edge; j++) {
        int cid = indices[edge_lb + j];
        DType attn_val = neigh_nodes_weight_off[j];
        if (i == 0 && tidx == 0) {
          attn_edge[hid * nnz + edge_lb + j] = attn_val / expAll;
        }
        if (pid < f)
          acc += attn_val * V[cid * hf + hid * f + pid];
      }
      // handle the node with no neighbor
      if (pid < f)
        out_feat[curr_node * hf + hid * f + pid] =
            (expAll != 0) ? acc / expAll : 0;
    }
  }
}

template <typename DType>
__global__ void fused_gt_hyper_inference(const int m, const int h, const int f,
                                         const int *row, const int *indptr,
                                         const int *indices, const DType *val,
                                         const DType *Q, const DType *K,
                                         const DType *V, DType *out_feat) {
  // launch dim (32, 8) * (num_nodes/8, 1)

  const int bidx = blockIdx.x;
  const int hid = blockIdx.y;
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;

  // the node bound of this block
  const int blockSize = blockDim.y;
  const int blk_node_lb = blockSize * bidx;
  const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

  // the edge bound of this block
  const int blk_edge_lb = indptr[blk_node_lb];
  const int blk_edge_hb = indptr[blk_node_hb];

  // the num of edges in this block
  const int blk_num_edge = blk_edge_hb - blk_edge_lb;

  // init smem
  extern __shared__ DType smem[];
  DType *neigh_nodes_weight = smem; // [8, f]

  // SDDMM, edge parallel
  int nnz_per_warp = (blk_num_edge + blockSize - 1) / blockSize;

  const int *rowoff = row + blk_edge_lb;
  const int *indicesoff = indices + blk_edge_lb;
  const DType *valoff = val + blk_edge_lb;

  int src;
  int dst;
  for (int i = 0; i < nnz_per_warp; i++) {
    int curr_edge = tidy * nnz_per_warp + i;
    // edge bound for curr block
    if (curr_edge < blk_num_edge) {
      src = __ldg(rowoff + curr_edge);
      dst = __ldg(indicesoff + curr_edge);

      // // the Q feature of row node
      const DType *Qoff = Q + src * f * h + hid * f;
      // the K feature of col node
      const DType *Koff = K + dst * f * h + hid * f;

      DType att_val = 0;
      for (int j = tidx; j < f; j += 64) {
        // float2 Q2 = reinterpret_cast<const float2*>(Qoff)[j];
        // float2 K2 = reinterpret_cast<const float2*>(Koff)[j];
        // att_val += vecDot2<float2, float>(Q2, K2);
        att_val += Qoff[j] * Koff[j];
        if (j + 32 < f)
          att_val += Qoff[j + 32] * Koff[j + 32];
      }
#pragma unroll
      for (int offset = 16; offset > 0; offset /= 2)
        att_val += __shfl_down_sync(full_mask, att_val, offset);
      if (tidx == 0) {
        // TODO consider to move val into smem
        neigh_nodes_weight[curr_edge] = att_val * valoff[curr_edge];
      }
    }
  }
  __syncthreads();

  // Softmax+SPMM, node parallel
  int curr_node = blk_node_lb + tidy;
  if (curr_node < blk_node_hb) {
    const int edge_lb = indptr[curr_node];
    const int edge_hb = indptr[curr_node + 1];
    const int num_edge = edge_hb - edge_lb;

    DType weightMax = -1e38;
    const int hf = h * f;
    // const int hfid = hid * f + tidx;

    DType *neigh_nodes_weight_off =
        neigh_nodes_weight + (edge_lb - blk_edge_lb);

    int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
    for (int j = 0; j < loop; j++) {
      DType weight = -1e38;
      int pid = tidx + (j << 5);
      if (pid < num_edge) {
        weight = neigh_nodes_weight_off[pid];
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32), weight);
      }
      __syncwarp();
      weightMax = MAX(weight, weightMax);
    }

    // compute the sum of exp
    DType expAll = 0;
    for (int j = 0; j < loop; j++) {
      int pid = tidx + (j << 5); // node need to process in loop j
      DType exptmp = 0;
      if (pid < num_edge) {
        DType weight = neigh_nodes_weight_off[pid];
        exptmp = exp(weight - weightMax);
        neigh_nodes_weight_off[pid] = exptmp;
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }

    // compute the output
    for (int i = tidx; i < f; i += 64) {
      DType acc = 0;
      DType acc2 = 0;
      // int pid = tidx + (i << 5);
      for (int j = 0; j < num_edge; j++) {
        int cid = indices[edge_lb + j];
        DType attn_val = neigh_nodes_weight_off[j];
        acc += attn_val * V[cid * hf + hid * f + i];
        if (i + 32 < f) {
          acc2 += attn_val * V[cid * hf + hid * f + i + 32];
        }
      }
      // handle the node with no neighbor
      out_feat[curr_node * hf + hid * f + i] = (expAll != 0) ? acc / expAll : 0;
      if (i + 32 < f) {
        out_feat[curr_node * hf + hid * f + i + 32] =
            (expAll != 0) ? acc2 / expAll : 0;
      }
    }
  }
}

template <typename DType>
__global__ void fused_gt_hyper_inference_small_f(
    const int m, const int h, const int f, const int *row, const int *indptr,
    const int *indices, const DType *val, const DType *Q, const DType *K,
    const DType *V, DType *out_feat) {
  // launch dim (32, 8) * (num_nodes/8, 1)

  const int bidx = blockIdx.x;
  const int hid = blockIdx.y;
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;

  // the node bound of this block
  const int blockSize = blockDim.y;
  const int blk_node_lb = blockSize * bidx;
  const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

  // the edge bound of this block
  const int blk_edge_lb = indptr[blk_node_lb];
  const int blk_edge_hb = indptr[blk_node_hb];

  // the num of edges in this block
  const int blk_num_edge = blk_edge_hb - blk_edge_lb;

  // init smem
  extern __shared__ DType smem[];
  DType *neigh_nodes_weight = smem; // [8, f]

  // SDDMM, edge parallel
  int nnz_per_warp = (blk_num_edge + blockSize - 1) / blockSize;

  const int *rowoff = row + blk_edge_lb;
  const int *indicesoff = indices + blk_edge_lb;
  const DType *valoff = val + blk_edge_lb;

  int src;
  int dst;
  for (int i = 0; i < nnz_per_warp; i++) {
    int curr_edge = tidy * nnz_per_warp + i;
    // edge bound for curr block
    if (curr_edge < blk_num_edge) {
      src = __ldg(rowoff + curr_edge);
      dst = __ldg(indicesoff + curr_edge);

      // // the Q feature of row node
      const DType *Qoff = Q + src * f * h + hid * f;
      // the K feature of col node
      const DType *Koff = K + dst * f * h + hid * f;

      DType att_val = 0;
      if (tidx < f) {
        att_val += Qoff[tidx] * Koff[tidx];
      }
#pragma unroll
      for (int offset = 16; offset > 0; offset /= 2)
        att_val += __shfl_down_sync(full_mask, att_val, offset);
      if (tidx == 0) {
        neigh_nodes_weight[curr_edge] = att_val * valoff[curr_edge];
      }
    }
  }
  __syncthreads();

  // Softmax+SPMM, node parallel
  int curr_node = blk_node_lb + tidy;
  if (curr_node < blk_node_hb) {
    const int edge_lb = indptr[curr_node];
    const int edge_hb = indptr[curr_node + 1];
    const int num_edge = edge_hb - edge_lb;

    DType weightMax = -1e38;
    const int hf = h * f;
    // const int hfid = hid * f + tidx;

    DType *neigh_nodes_weight_off =
        neigh_nodes_weight + (edge_lb - blk_edge_lb);

    int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
    for (int j = 0; j < loop; j++) {
      DType weight = -1e38;
      int pid = tidx + (j << 5);
      if (pid < num_edge) {
        weight = neigh_nodes_weight_off[pid];
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32), weight);
      }
      __syncwarp();
      weightMax = MAX(weight, weightMax);
    }

    // compute the sum of exp
    DType expAll = 0;
    for (int j = 0; j < loop; j++) {
      int pid = tidx + (j << 5); // node need to process in loop j
      DType exptmp = 0;
      if (pid < num_edge) {
        DType weight = neigh_nodes_weight_off[pid];
        exptmp = exp(weight - weightMax);
        neigh_nodes_weight_off[pid] = exptmp;
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }

    // compute the output
    DType acc = 0;
    for (int j = 0; j < num_edge; j++) {
      int cid = indices[edge_lb + j];
      DType attn_val = neigh_nodes_weight_off[j];
      if (tidx < f)
        acc += attn_val * V[cid * hf + hid * f + tidx];
    }
    // handle the node with no neighbor
    if (tidx < f)
      out_feat[curr_node * hf + hid * f + tidx] =
          (expAll != 0) ? acc / expAll : 0;
  }
}

template <typename DType>
__global__ void fused_inference_kernel_hyper_row_switch(
    const int m, const int h, const int f, const int *row, const int *indptr,
    const int *indices, const DType *val, const DType *Q, const DType *K,
    const DType *V, DType *out_feat) {
  // launch dim (32, 8) * (num_nodes/8, 1)
  const int bidx = blockIdx.x;
  const int hid = blockIdx.y;
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;

  // the node bound of this block
  const int blockSize = blockDim.y;
  const int blk_node_lb = blockSize * bidx;
  const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

  // the edge bound of this block
  const int blk_edge_lb = indptr[blk_node_lb];
  const int blk_edge_hb = indptr[blk_node_hb];

  // the num of edges in this block
  const int blk_num_edge = blk_edge_hb - blk_edge_lb;

  // init smem
  extern __shared__ DType smem[];
  DType *neigh_nodes_weight = smem; // [8, f]

  float Q_row[32];

  // SDDMM, edge parallel
  int nnz_per_warp = (blk_num_edge + blockSize - 1) / blockSize;

  const int *rowoff = row + blk_edge_lb;
  const int *indicesoff = indices + blk_edge_lb;
  const DType *valoff = val + blk_edge_lb;
  // DType *Q_smemoff = Q_smem + tidy * f;

  int src_old = -1;
  int src;
  int dst;
  for (int i = 0; i < nnz_per_warp; i++) {
    int curr_edge = tidy * nnz_per_warp + i;
    // edge bound for curr block
    if (curr_edge < blk_num_edge) {
      src = __ldg(rowoff + curr_edge);
      dst = __ldg(indicesoff + curr_edge);
      if (src != src_old) {
        src_old = src;
        for (int j = tidx; j < f; j += 64) {
          int pid = j / WARP_SIZE;
          Q_row[pid] = Q[src_old * f * h + hid * f + j];
          if (j + 32 < f) {
            Q_row[pid + 1] = Q[src_old * f * h + hid * f + j + 32];
          }
        }
      }
      // the K feature of col node
      const DType *Koff = K + dst * f * h + hid * f;
      DType att_val = 0;
      for (int j = tidx; j < f; j += 64) {
        int idx = j / WARP_SIZE;
        att_val += Q_row[idx] * Koff[j];
        if (j + 32 < f)
          att_val += Q_row[idx + 1] * Koff[j + 32];
      }
#pragma unroll
      for (int offset = 16; offset > 0; offset /= 2)
        att_val += __shfl_down_sync(full_mask, att_val, offset);
      if (tidx == 0) {
        // TODO consider to move val into smem
        neigh_nodes_weight[curr_edge] = att_val * valoff[curr_edge];
      }
    }
  }
  __syncthreads();

  // Softmax+SPMM, node parallel
  int curr_node = blk_node_lb + tidy;
  if (curr_node < blk_node_hb) {
    const int edge_lb = indptr[curr_node];
    const int edge_hb = indptr[curr_node + 1];
    const int num_edge = edge_hb - edge_lb;

    DType weightMax = -1e38;
    const int hf = h * f;
    // const int hfid = hid * f + tidx;

    DType *neigh_nodes_weight_off =
        neigh_nodes_weight + (edge_lb - blk_edge_lb);

    int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
    for (int j = 0; j < loop; j++) {
      DType weight = -1e38;
      int pid = tidx + (j << 5);
      if (pid < num_edge) {
        weight = neigh_nodes_weight_off[pid];
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32), weight);
      }
      __syncwarp();
      weightMax = MAX(weight, weightMax);
    }

    // compute the sum of exp
    DType expAll = 0;
    for (int j = 0; j < loop; j++) {
      int pid = tidx + (j << 5); // node need to process in loop j
      DType exptmp = 0;
      if (pid < num_edge) {
        DType weight = neigh_nodes_weight_off[pid];
        exptmp = exp(weight - weightMax);
        neigh_nodes_weight_off[pid] = exptmp;
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }

    // compute the output
    int loop_f = (f + WARP_SIZE - 1) / WARP_SIZE;
    for (int i = 0; i < loop_f; i++) {
      DType acc = 0;
      int pid = tidx + (i << 5);
      for (int j = 0; j < num_edge; j++) {
        int cid = indices[edge_lb + j];
        DType attn_val = neigh_nodes_weight_off[j];
        if (pid < f)
          acc += attn_val * V[cid * hf + hid * f + pid];
      }
      // handle the node with no neighbor
      if (pid < f)
        out_feat[curr_node * hf + hid * f + pid] =
            (expAll != 0) ? acc / expAll : 0;
    }
  }
}

// csc format spmm kernel
// efeat (nnz, h, efeat_len), ufeat (n, h, ufeat_len), in csr format
template <typename DType>
__global__ void
spmm_csc_kernel(int n, int ufeat_len, int efeat_len, int out_len,
                const int *indptr, const int *indices, const int *val_idx,
                const DType *ufeat, const DType *efeat, DType *out) {
  int hid = blockIdx.y;
  int ty = blockIdx.x * blockDim.y + threadIdx.y; // 0-n
  const int stride_x = blockDim.x * gridDim.y;    // f*h

  if (ty < n) {
    int tx = blockIdx.y * blockDim.x + threadIdx.x; // 0-f
    while (tx < out_len) {
      DType acc = 0;
      for (int i = indptr[ty]; i < indptr[ty + 1]; ++i) {
        const int eid = val_idx[i];
        const int cid = __ldg(indices + i);
        const DType *uoff = ufeat + cid * ufeat_len;
        const DType *eoff = efeat + hid * efeat_len;
        DType tmp_out = uoff[tx] * eoff[eid];
        acc += tmp_out;
      }
      out[ty * out_len + tx] = acc;
      tx += stride_x;
    }
  }
}

std::vector<torch::Tensor>
gt_hyper_inference_cuda(torch::Tensor indptr, torch::Tensor indices,
                        torch::Tensor rows, torch::Tensor val, int smem_consume,
                        torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
  // Q: torch.Size([6248, 10, 8])
  const auto m = indptr.size(0) - 1; // num of nodes
  const auto nnz = indices.size(0);  // num of edges
  const auto h = Q.size(1);          // num of heads
  const auto f = Q.size(2);          // num of feats
  const auto val_size = val.size(0); // check if val is scalar
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);

  const int ntx = 32;
  const int nty = 8;

  const int nbx = (m + nty - 1) / nty;
  const int nby = h;
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);
  const int smem_size = smem_consume * sizeof(float);

  if (f <= 32) {
    CUDA_KERNEL_CALL(
        (fused_gt_hyper_inference_small_f<float>), nblks, nthrs, smem_size, m,
        h, f, rows.data_ptr<int>(), indptr.data_ptr<int>(),
        indices.data_ptr<int>(), val.data_ptr<float>(), Q.data_ptr<float>(),
        K.data_ptr<float>(), V.data_ptr<float>(), out_feat.data_ptr<float>());
  } else {
    CUDA_KERNEL_CALL((fused_gt_hyper_inference<float>), nblks, nthrs, smem_size,
                     m, h, f, rows.data_ptr<int>(), indptr.data_ptr<int>(),
                     indices.data_ptr<int>(), val.data_ptr<float>(),
                     Q.data_ptr<float>(), K.data_ptr<float>(),
                     V.data_ptr<float>(), out_feat.data_ptr<float>());
  }

  return {out_feat};
}

std::vector<torch::Tensor>
gt_hyper_forward_cuda(torch::Tensor row_ptr, torch::Tensor col_ind,
                      torch::Tensor rows, torch::Tensor val,
                      torch::Tensor col_ptr, torch::Tensor row_ind,
                      torch::Tensor val_idx, int smem_consume, torch::Tensor Q,
                      torch::Tensor K, torch::Tensor V) {
  // Q: torch.Size([6248, 10, 8])
  const auto m = row_ptr.size(0) - 1; // num of nodes
  const auto nnz = col_ind.size(0);   // num of edges
  const auto h = Q.size(1);           // num of heads
  const auto f = Q.size(2);           // num of feats
  auto devid = row_ptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  auto attn_edge = torch::empty({h, nnz}, options);

  const int ntx = 32;
  const int nty = 8;

  const int nbx = (m + nty - 1) / nty;
  const int nby = h;
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);
  const int smem_size = smem_consume * sizeof(float);

  CUDA_KERNEL_CALL((fused_gt_hyper<float>), nblks, nthrs, smem_size, m, nnz, h,
                   f, rows.data_ptr<int>(), row_ptr.data_ptr<int>(),
                   col_ind.data_ptr<int>(), val.data_ptr<float>(),
                   Q.data_ptr<float>(), K.data_ptr<float>(),
                   V.data_ptr<float>(), attn_edge.data_ptr<float>(),
                   out_feat.data_ptr<float>());

  return {out_feat, attn_edge};
}