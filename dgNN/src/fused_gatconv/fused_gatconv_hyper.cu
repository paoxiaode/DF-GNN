#include "hip/hip_runtime.h"
#include "../util/computeUtil.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>

template <typename DType>
__global__ void fused_gat_hyper_inference(
    int m, int h, int f, const DType *attn_row, const DType *attn_col,
    const int *row, const int *indptr, const int *indices, const DType *in_feat,
    const DType negative_slope, DType *out_feat) {
  // launch dim (32, 8) * (num_nodes/8, 1)
  const int bidx = blockIdx.x;
  const int hid = blockIdx.y;
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int tid = tidy * 32 + tidx;

  // the node bound of this block
  const int blockSize = blockDim.y;
  const int blk_node_lb = blockSize * bidx;
  const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

  // the edge bound of this block
  const int blk_edge_lb = indptr[blk_node_lb];
  const int blk_edge_hb = indptr[blk_node_hb];

  // the num of edges in this block
  const int blk_num_edge = blk_edge_hb - blk_edge_lb;

  // init smem
  extern __shared__ DType smem[];
  DType *neigh_nodes_weight = smem; // [8, f]

  const int *rowoff = row + blk_edge_lb;
  const int *indicesoff = indices + blk_edge_lb;

  // SDDMM, edge parallel
  for (int i = tid; i < blk_num_edge; i += blockSize * WARP_SIZE) {
    if (i < blk_num_edge) {
      const int src = __ldg(rowoff + i);
      const int dst = __ldg(indicesoff + i);
      DType weight = attn_row[src * h + hid] + attn_col[dst * h + hid];
      weight = LeakyRelu(weight, negative_slope);
      neigh_nodes_weight[i] = weight;
    }
  }
  __syncthreads();

  // Softmax+SPMM, node parallel
  int curr_node = blk_node_lb + tidy;
  if (curr_node < blk_node_hb) {
    const int edge_lb = indptr[curr_node];
    const int edge_hb = indptr[curr_node + 1];
    const int num_edge = edge_hb - edge_lb;

    DType weightMax = -1e38;
    const int hf = h * f;
    // const int hfid = hid * f + tidx;

    DType *neigh_nodes_weight_off =
        neigh_nodes_weight + (edge_lb - blk_edge_lb);

    int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
    for (int j = 0; j < loop; j++) {
      DType weight = -1e38;
      int pid = tidx + (j << 5);
      if (pid < num_edge) {
        weight = neigh_nodes_weight_off[pid];
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32), weight);
      }
      __syncwarp();
      weightMax = MAX(weight, weightMax);
    }

    // compute the sum of exp
    DType expAll = 0;
    for (int j = 0; j < loop; j++) {
      int pid = tidx + (j << 5); // node need to process in loop j
      DType exptmp = 0;
      if (pid < num_edge) {
        DType weight = neigh_nodes_weight_off[pid];
        exptmp = exp(weight - weightMax);
        neigh_nodes_weight_off[pid] = exptmp;
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }
    expAll = (expAll != 0) ? 1.0f / expAll : 0;

    // compute the output
    int loop_f = (f + WARP_SIZE - 1) / WARP_SIZE;
    for (int i = 0; i < loop_f; i++) {
      DType acc = 0;
      int pid = tidx + (i << 5);
      for (int j = 0; j < num_edge; j++) {
        int cid = indices[edge_lb + j];
        DType attn_val = neigh_nodes_weight_off[j];
        if (pid < f)
          acc += attn_val * in_feat[cid * hf + hid * f + pid];
      }
      // handle the node with no neighbor
      if (pid < f)
        out_feat[curr_node * hf + hid * f + pid] = acc * expAll;
    }
  }
}

template <typename DType>
__global__ void fused_gat_hyper_inference_vec4(
    int m, int h, int f, const DType *attn_row, const DType *attn_col,
    const int *row, const int *indptr, const int *indices, const DType *in_feat,
    const DType negative_slope, DType *out_feat) {
  // launch dim (32, 8) * (num_nodes/8, 1)
  const int bidx = blockIdx.x;
  const int hid = blockIdx.y;
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int tid = tidy * 32 + tidx;

  // the node bound of this block
  const int blockSize = blockDim.y;
  const int blk_node_lb = blockSize * bidx;
  const int blk_node_hb = MIN(blk_node_lb + blockSize, m);

  // the edge bound of this block
  const int blk_edge_lb = indptr[blk_node_lb];
  const int blk_edge_hb = indptr[blk_node_hb];

  // the num of edges in this block
  const int blk_num_edge = blk_edge_hb - blk_edge_lb;

  // init smem
  extern __shared__ DType smem[];
  DType *neigh_nodes_weight = smem; // [8, f]

  const int *rowoff = row + blk_edge_lb;
  const int *indicesoff = indices + blk_edge_lb;

  // SDDMM, edge parallel
  for (int i = tid; i < blk_num_edge; i += blockSize * WARP_SIZE) {
    if (i < blk_num_edge) {
      const int src = __ldg(rowoff + i);
      const int dst = __ldg(indicesoff + i);
      DType weight = attn_row[src * h + hid] + attn_col[dst * h + hid];
      weight = LeakyRelu(weight, negative_slope);
      neigh_nodes_weight[i] = weight;
    }
  }
  __syncthreads();

  // Softmax+SPMM, node parallel
  int curr_node = blk_node_lb + tidy;
  if (curr_node < blk_node_hb) {
    const int edge_lb = indptr[curr_node];
    const int edge_hb = indptr[curr_node + 1];
    const int num_edge = edge_hb - edge_lb;

    DType weightMax = -1e38;
    const int hf = h * f;
    // const int hfid = hid * f + tidx;

    DType *neigh_nodes_weight_off =
        neigh_nodes_weight + (edge_lb - blk_edge_lb);

    int loop = (num_edge + WARP_SIZE - 1) / WARP_SIZE;
    for (int j = 0; j < loop; j++) {
      DType weight = -1e38;
      int pid = tidx + (j << 5);
      if (pid < num_edge) {
        weight = neigh_nodes_weight_off[pid];
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        weight = max(__shfl_xor_sync(0xffffffff, weight, stride, 32), weight);
      }
      __syncwarp();
      weightMax = MAX(weight, weightMax);
    }

    // compute the sum of exp
    DType expAll = 0;
    for (int j = 0; j < loop; j++) {
      int pid = tidx + (j << 5); // node need to process in loop j
      DType exptmp = 0;
      if (pid < num_edge) {
        DType weight = neigh_nodes_weight_off[pid];
        exptmp = exp(weight - weightMax);
        neigh_nodes_weight_off[pid] = exptmp;
      }
      __syncwarp();
#pragma unroll
      for (int stride = 16; stride > 0; stride >>= 1) {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }
    expAll = (expAll != 0) ? 1.0f / expAll : 0;

    DType *Outoff = out_feat + curr_node * h * f + hid * f;
    for (int i = tidx; i < f / 4; i += 32) {
      // DType acc = 0;
      DType acc[4] = {0, 0, 0, 0};
      for (int j = 0; j < num_edge; j++) {
        int cid = indices[edge_lb + j];
        DType attn_val = neigh_nodes_weight_off[j];
        const DType *featoff = in_feat + cid * h * f + hid * f + 4 * i;
        Mul4_const<float>(acc, featoff, attn_val);
      }
      // handle the node with no neighbor
      selfMulConst4<float>(acc, expAll);
      Store<float4, float>(Outoff, acc, 4 * i);
    }
  }
}

void gat_hyper_inference_launch(int m, int nnz, int h, int f, int smem_consume,
                                const float *attn_row, const float *attn_col,
                                const int *indptr, const int *indices,
                                const int *rows, float negative_slope,
                                const float *in_feat, float *out_feat) {
  const int ntx = 32;
  const int nty = 8;

  const int nbx = (m + nty - 1) / nty;
  const int nby = h;
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);
  const int smem_size = smem_consume * sizeof(float);

  if ((f % 128) == 0) {
    CUDA_KERNEL_CALL((fused_gat_hyper_inference_vec4<float>), nblks, nthrs,
                     smem_size, m, h, f, attn_row, attn_col, rows, indptr,
                     indices, in_feat, negative_slope, out_feat);
  } else {
    CUDA_KERNEL_CALL((fused_gat_hyper_inference<float>), nblks, nthrs,
                     smem_size, m, h, f, attn_row, attn_col, rows, indptr,
                     indices, in_feat, negative_slope, out_feat);
  }
}

torch::Tensor gat_hyper_inference_cuda(int smem_consume, torch::Tensor attn_row,
                                       torch::Tensor attn_col,
                                       torch::Tensor indptr,
                                       torch::Tensor indices,
                                       torch::Tensor rows, float negative_slope,
                                       torch::Tensor in_feat) {
  const auto m = indptr.size(0) - 1;
  const auto nnz = indices.size(0);
  const auto h = attn_row.size(1);
  const auto f = in_feat.size(2);
  auto devid = attn_row.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::empty({m, h, f}, options);
  // printf("gat_inference\n");
  gat_hyper_inference_launch(
      m, nnz, h, f, smem_consume, attn_row.data_ptr<float>(),
      attn_col.data_ptr<float>(), indptr.data_ptr<int>(),
      indices.data_ptr<int>(), rows.data_ptr<int>(), negative_slope,
      in_feat.data_ptr<float>(), out_feat.data_ptr<float>());
  return out_feat;
}