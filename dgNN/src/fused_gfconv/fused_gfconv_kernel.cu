#include "hip/hip_runtime.h"
#include "../util/computeUtil.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>
#include <unistd.h>
#include <stdio.h>

using namespace std;

extern "C" bool isPow2(unsigned int x)
{
  return ((x & (x - 1)) == 0 && x >= 32);
}

#define CUDA_CALL(func)                                      \
  {                                                          \
    hipError_t e = (func);                                  \
    CHECK(e == hipSuccess || e == hipErrorDeinitialized) \
        << "CUDA: " << hipGetErrorString(e);                \
  }

#define CUSPARSE_CALL(func)                                         \
  {                                                                 \
    hipsparseStatus_t e = (func);                                    \
    CHECK(e == HIPSPARSE_STATUS_SUCCESS) << "CUSPARSE ERROR: " << e; \
  }

#define CUDA_KERNEL_CALL(kernel, nblks, nthrs, shmem, ...)          \
  {                                                                 \
    {                                                               \
      (kernel)<<<(nblks), (nthrs), (shmem)>>>(__VA_ARGS__);         \
      hipError_t e = hipGetLastError();                           \
      CHECK(e == hipSuccess || e == hipErrorDeinitialized)      \
          << "CUDA kernel launch error: " << hipGetErrorString(e); \
    }                                                               \
  }

__device__ __forceinline__ float warpReduceSum(float sum, int blockSize)
{
  if (blockSize >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16);
  if (blockSize >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8);
  if (blockSize >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4);
  if (blockSize >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2);
  if (blockSize >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1);
  return sum;
}

struct Dot
{
  static __device__ __forceinline__ float
  Call(const float *lhs, const float *rhs, int len = 1)
  {
    float rst = static_cast<float>(0.0f);
    for (int i = 0; i < len; ++i)
    {
      rst += lhs[i] * rhs[i];
    }
    return rst;
  }
};

template <typename DType>
__global__ void sddmmCooKernel(const int lhs_len, const int rhs_len, const int out_len,
                               const int nnz, const int reduce_size,
                               const int *row, const int *col, const float *data,
                               const float *lhs, const float *rhs, float *out)
{
  int ty = blockIdx.x * blockDim.y + threadIdx.y;
  if (ty < nnz)
  {
    const int src = __ldg(row + ty);
    const int dst = __ldg(col + ty);
    const int eid = ty;
    const DType *lhsoff = lhs + src * lhs_len;
    const DType *rhsoff = rhs + dst * rhs_len;
    DType *outoff = out + eid * out_len;
    int tx = threadIdx.x; // tx < 32
    for (int i = blockIdx.y; i < out_len; i += gridDim.y)
    { // over output feature dimension
      DType val = 0;
      for (int j = tx; j < reduce_size; j += 64)
      {
        val += lhsoff[i * reduce_size + j] *
               rhsoff[i * reduce_size + j];
        if (j + 32 < reduce_size)
          val += lhsoff[i * reduce_size + j + 32] *
                 rhsoff[i * reduce_size + j + 32];
      }
#pragma unroll
      for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(full_mask, val, offset);
      if (tx == 0){
        outoff[i] = val;
        // printf("%f\n",val );
      }
    }
  }
}


__global__ void sddmmCsrKernel(const int m, const int nnz, const int h, const int f,
                                          const int *indptr, const int *indices, const float *val,
                                          const float *Q, const float *K, float *attn_edge)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int num_neighbor = hb - lb;
  static __shared__ float warpLevelSums[WARP_SIZE];
  const int hf = h * f;
  const int hfid = hid * f + fid;
  const int laneId = fid % WARP_SIZE;
  const int warpId = fid / WARP_SIZE;
  const int blockSize = blockDim.x * blockDim.y;

  float Q_i = Q[rid * hf + hfid];

  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    float weight_partial = 0;

    int cid = indices[lb + j];
    weight_partial = Q_i * K[cid * hf + hfid];

    __syncthreads();
    weight_partial = warpReduceSum(weight_partial, blockSize);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
    if (fid == 0)
    {
      attn_edge[lb+j] = weight_partial * val[lb + j];
    }
  }
  __syncthreads();
}

__global__ void softMax_SPMM(const int m, const int nnz, const int h, const int f,
                             const int *indptr, const int *indices, const float *val,
                             const float *V, const float *attn_edge,
                             float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *neigh_nodes_weight = smem;
  float weightMax = -1e38;
  const int hf = h * f;
  const int hfid = hid * f + fid;

  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    if (fid == 0)
    {
      neigh_nodes_weight[j] = attn_edge[lb + j];
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  __syncthreads();

  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = indices[lb + j];
    float weight = neigh_nodes_weight[j];
    attn_val = exp(weight - weightMax) / expAll;
    acc += attn_val * V[cid * hf + hfid];
    __syncthreads();
  }

  out_feat[rid * hf + hfid] = acc;
}

template <unsigned int blockSize>
__global__ void fused_forward_kernel_pow2(const int m, const int nnz, const int h, const int f,
                                          const int *indptr, const int *indices, const float *val,
                                          const float *Q, const float *K, const float *V,
                                          float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *neigh_nodes_weight = smem;
  float weightMax = -1e38;
  static __shared__ float warpLevelSums[WARP_SIZE];
  const int hf = h * f;
  const int hfid = hid * f + fid;
  const int laneId = fid % WARP_SIZE;
  const int warpId = fid / WARP_SIZE;
  float Q_i = Q[rid * hf + hfid];

  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    float weight_partial = 0;

    int cid = indices[lb + j];
    weight_partial = Q_i * K[cid * hf + hfid];

    __syncthreads();
    weight_partial = warpReduceSum(weight_partial, blockSize);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
    if (fid == 0)
    {
      neigh_nodes_weight[j] = weight_partial * val[lb + j];
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  __syncthreads();

  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = indices[lb + j];
    float weight = neigh_nodes_weight[j];
    attn_val = exp(weight - weightMax) / expAll;
    acc += attn_val * V[cid * hf + hfid];
    __syncthreads();
  }

  out_feat[rid * hf + hfid] = acc;
}

__global__ void fused_forward_kernel(const int m, const int nnz, const int h, const int f,
                                     const int *indptr, const int *indices, const float *val,
                                     const float *Q, const float *K, const float *V,
                                     float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int threads_x = blockDim.x; // 32
  const int threads_y = blockDim.y; // f/32
  const int blockSize = threads_x * threads_y;
  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *curr_node_feature = smem;
  float *neigh_nodes_weight = (float *)&curr_node_feature[f];
  float weightMax = -1e38;
  // init the shared memory
  float Q_i = 0;
  if (fid < f)
  {
    Q_i = Q[rid * h * f + hid * f + fid];
    // curr_node_feature[fid] = Q[rid * h * f + hid * f + fid];
  }

  // compute the attention weight
  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    float weight_partial = 0;
    if (fid < f)
    {
      int cid = indices[lb + j];
      weight_partial = Q_i * K[cid * h * f + hid * f + fid];
    }
    __syncthreads();
    static __shared__ float warpLevelSums[WARP_SIZE];
    const int laneId = fid % WARP_SIZE;
    const int warpId = fid / WARP_SIZE;
    weight_partial = warpReduceSum(weight_partial, blockSize);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
    if (fid == 0)
    {
      neigh_nodes_weight[j] = weight_partial;
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  __syncthreads();

  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = indices[lb + j];
    if (fid < f)
    {
      float weight = neigh_nodes_weight[j];
      attn_val = exp(weight - weightMax) / expAll;
      acc += attn_val * V[cid * h * f + hid * f + fid];
    }
    __syncthreads();
  }
  if (fid < f)
    out_feat[rid * h * f + hid * f + fid] = acc;
}

__global__ void fused_forward_ell_kernel(const int m, const int nnz, const int h, const int f,
                                         const int *indptr, const int *indices,
                                         const int *row_index, const int *rows_per_tb, const float *val,
                                         const float *Q, const float *K, const float *V,
                                         float *out_feat)
{
  const int bid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim
  const int lrow = rows_per_tb[bid];              // row rid elements
  const int hrow = rows_per_tb[bid + 1];
  const int num_rows = hrow - lrow;
  extern __shared__ float smem[];
  float *curr_node_feature = smem;
  float *neigh_nodes_weight = (float *)&curr_node_feature[f];

  const int threads_x = blockDim.x; // 32
  const int threads_y = blockDim.y; // f/32
  const int blockSize = threads_x * threads_y;
  float weightMax = -1e38;

  for (int row = 0; row < num_rows; row++)
  {
    int rid = row_index[lrow + row];
    const int lb = indptr[rid]; // row rid elements
    const int hb = indptr[rid + 1];

    const int num_neighbor = hb - lb;

    // init the shared memory
    if (fid < f)
    {
      curr_node_feature[fid] = Q[rid * h * f + hid * f + fid];
    }

    // compute the attention weight
    for (int j = 0; j < num_neighbor; j++)
    {
      float weight = 0;
      float weight_partial = 0;
      if (fid < f)
      {
        int cid = indices[lb + j];
        weight_partial = curr_node_feature[fid] * K[cid * h * f + hid * f + fid];
      }
      __syncthreads();
      static __shared__ float warpLevelSums[WARP_SIZE];
      const int laneId = fid % WARP_SIZE;
      const int warpId = fid / WARP_SIZE;
      weight_partial = warpReduceSum(weight_partial, blockSize);
      if (laneId == 0)
        warpLevelSums[warpId] = weight_partial;
      __syncthreads();
      weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
      if (warpId == 0)
        weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
      if (fid == 0)
      {
        neigh_nodes_weight[j] = weight_partial;
      }
      __syncthreads();
      weight = neigh_nodes_weight[j];
      weightMax = MAX(weight, weightMax);
    }
    __syncthreads();

    // compute the sum of exp
    int loop = (num_neighbor + 31) / 32;
    float expAll = 0;
    for (int j = 0; j < loop; j++)
    {
      int pid = threadIdx.x + (j << 5); // node need to process in loop j
      float exptmp = 0;
      if (pid < num_neighbor)
      {
        float weight = neigh_nodes_weight[pid];
        exptmp = exp(weight - weightMax);
      }
      __syncwarp();
      for (int stride = 16; stride > 0; stride >>= 1)
      {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }
    __syncthreads();

    // compute the output
    float acc = 0;
    for (int j = 0; j < num_neighbor; j++)
    {
      float attn_val;
      int cid = indices[lb + j];
      if (fid < f)
      {
        float weight = neigh_nodes_weight[j];
        attn_val = exp(weight - weightMax) / expAll;
        acc += attn_val * V[cid * h * f + hid * f + fid];
      }
      __syncthreads();
    }
    if (fid < f)
      out_feat[rid * h * f + hid * f + fid] = acc;
    __syncthreads();
  }
}

void gf_forward(int m, int nnz, int h, int f,
                const int *indptr, const int *indices, const float *val,
                const float *Q, const float *K, const float *V,
                float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  const dim3 nblks(m, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (fused_forward_kernel),
      nblks, nthrs, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
      Q, K, V, out_feat);
  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // float elapsedTime;
  // hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("Time of fused kernel: %f \n", elapsedTime);
}

void gf_forward_nofuse(int m, int nnz, int h, int f,
                       const int *indptr, const int *indices, const int *rows, const float *val,
                       const float *Q, const float *K, const float *V,
                       float *attn_edge, float *out_feat)
{
  // const int ntx = 32; // on feature dimension
  // const int nty = 8;  // on out dimension
  // const int nbx = (nnz + nty - 1) / nty;
  // const int nby = FindNumBlocks<'y'>(h);
  // const dim3 nblks(nbx, nby);
  // const dim3 nthrs(ntx, nty);

  // CUDA_KERNEL_CALL(
  //     (sddmmCooKernel<float>),
  //     nblks, nthrs, 0, f * h, f * h, h, nnz, f, rows, indices, val,
  //     Q, K, attn_edge);

  const dim3 nblks2(m, h, 1);
  const dim3 nthrs2(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (sddmmCsrKernel),
      nblks2, nthrs2, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
      Q, K, attn_edge);

  // const dim3 nblks2(m, h, 1);
  // const dim3 nthrs2(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (softMax_SPMM),
      nblks2, nthrs2, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
      V, attn_edge, out_feat);
}

void gf_forward_pow2(int m, int nnz, int h, int f,
                     const int *indptr, const int *indices, const float *val,
                     const float *Q, const float *K, const float *V,
                     float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);
  const dim3 nblks(m, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  switch (f)
  {
  case 4096:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<4096>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;
  case 2048:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<2048>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;
  case 1024:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<1024>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;
  case 512:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<512>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;

  case 256:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<256>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;

  case 128:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<128>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;

  case 64:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<64>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;

  case 32:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<32>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
        Q, K, V, out_feat);
    break;
  }
}

void gf_ell_forward(int m, int nnz, int h, int f, int num_tb,
                    const int *indptr, const int *indices,
                    const int *row_index, const int *rows_per_tb, const float *val,
                    const float *Q, const float *K, const float *V,
                    float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  const dim3 nblks(num_tb, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (fused_forward_ell_kernel),
      nblks, nthrs, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, row_index, rows_per_tb, val,
      Q, K, V, out_feat);
  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // float elapsedTime;
  // hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("Time of fused kernel: %f \n", elapsedTime);
}

std::vector<torch::Tensor>
gf_forward_cuda(torch::Tensor indptr,
                torch::Tensor indices,
                torch::Tensor val, torch::Tensor Q,
                torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = indptr.size(0) - 1; // num of nodes
  const auto nnz = indices.size(0);  // num of edges
  const auto h = Q.size(1);          // num of heads
  const auto f = Q.size(2);          // num of feats
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  if (isPow2(f))
  {
    gf_forward_pow2(m, nnz, h, f,
                    indptr.data_ptr<int>(), indices.data_ptr<int>(), val.data_ptr<float>(),
                    Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                    out_feat.data_ptr<float>());
  }
  else
  {
    gf_forward(m, nnz, h, f,
               indptr.data_ptr<int>(), indices.data_ptr<int>(), val.data_ptr<float>(),
               Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
               out_feat.data_ptr<float>());
  }
  return {out_feat};
}

std::vector<torch::Tensor>
gf_hyper_forward_cuda(torch::Tensor indptr,
                      torch::Tensor indices, torch::Tensor rows,
                      torch::Tensor val, torch::Tensor Q,
                      torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = indptr.size(0) - 1; // num of nodes
  const auto nnz = indices.size(0);  // num of edges
  const auto h = Q.size(1);          // num of heads
  const auto f = Q.size(2);          // num of feats
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  auto attn_edge = torch::zeros({nnz * h}, options);
  gf_forward_nofuse(m, nnz, h, f,
                    indptr.data_ptr<int>(), indices.data_ptr<int>(), rows.data_ptr<int>(), val.data_ptr<float>(),
                    Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                    attn_edge.data_ptr<float>(), out_feat.data_ptr<float>());
  
  return {out_feat};
}

std::vector<torch::Tensor>
gf_ell_forward_cuda(torch::Tensor indptr,
                    torch::Tensor indices,
                    torch::Tensor row_index,
                    torch::Tensor rows_per_tb,
                    torch::Tensor val, torch::Tensor Q,
                    torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = indptr.size(0) - 1;           // num of nodes
  const auto nnz = indices.size(0);            // num of edges
  const auto h = Q.size(1);                    // num of heads
  const auto f = Q.size(2);                    // num of feats
  const auto num_tb = rows_per_tb.size(0) - 1; // num of thread blocks
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  gf_ell_forward(m, nnz, h, f, num_tb,
                 indptr.data_ptr<int>(), indices.data_ptr<int>(),
                 row_index.data_ptr<int>(), rows_per_tb.data_ptr<int>(), val.data_ptr<float>(),
                 Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                 out_feat.data_ptr<float>());
  return {out_feat};
}