#include "hip/hip_runtime.h"
#include "../util/computeUtil.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>
#include <unistd.h>
#include <stdio.h>

using namespace std;

extern "C" bool isMul32(int x)
{
  return (x >= 0 && x % 32 == 0);
}

#define CUDA_CALL(func)                                      \
  {                                                          \
    hipError_t e = (func);                                  \
    CHECK(e == hipSuccess || e == hipErrorDeinitialized) \
        << "CUDA: " << hipGetErrorString(e);                \
  }

#define CUSPARSE_CALL(func)                                         \
  {                                                                 \
    hipsparseStatus_t e = (func);                                    \
    CHECK(e == HIPSPARSE_STATUS_SUCCESS) << "CUSPARSE ERROR: " << e; \
  }

#define CUDA_KERNEL_CALL(kernel, nblks, nthrs, shmem, ...)          \
  {                                                                 \
    {                                                               \
      (kernel)<<<(nblks), (nthrs), (shmem)>>>(__VA_ARGS__);         \
      hipError_t e = hipGetLastError();                           \
      CHECK(e == hipSuccess || e == hipErrorDeinitialized)      \
          << "CUDA kernel launch error: " << hipGetErrorString(e); \
    }                                                               \
  }

__device__ __forceinline__ float warpReduceSum(float sum, int blockSize)
{
  if (blockSize >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16);
  if (blockSize >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8);
  if (blockSize >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4);
  if (blockSize >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2);
  if (blockSize >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1);
  return sum;
}

template <typename DType>
__global__ void sddmmCooKernel(const int lhs_len, const int rhs_len, const int out_len,
                               const int nnz, const int reduce_size,
                               const int *row, const int *col, const DType *data,
                               const DType *lhs, const DType *rhs, DType *out)
{
  int ty = blockIdx.x * blockDim.y + threadIdx.y;
  if (ty < nnz)
  {
    const int src = __ldg(row + ty);
    const int dst = __ldg(col + ty);
    const int eid = ty;
    const DType *lhsoff = lhs + src * lhs_len;
    const DType *rhsoff = rhs + dst * rhs_len;
    DType *outoff = out + eid * out_len;
    int tx = threadIdx.x; // tx < 32
    for (int i = blockIdx.y; i < out_len; i += gridDim.y)
    { // over output feature dimension
      DType val = 0;
      for (int j = tx; j < reduce_size; j += 64)
      {
        val += lhsoff[i * reduce_size + j] *
               rhsoff[i * reduce_size + j];
        if (j + 32 < reduce_size)
          val += lhsoff[i * reduce_size + j + 32] *
                 rhsoff[i * reduce_size + j + 32];
      }
#pragma unroll
      for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(full_mask, val, offset);
      if (tx == 0)
      {
        outoff[i] = val;
      }
    }
  }
}

__global__ void sddmmCsrKernel(const int m, const int nnz, const int h, const int f,
                               const int *indptr, const int *indices, const float *val,
                               const float *Q, const float *K, float *attn_edge)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int num_neighbor = hb - lb;
  static __shared__ float warpLevelSums[WARP_SIZE];
  const int hf = h * f;
  const int hfid = hid * f + fid;
  const int laneId = fid % WARP_SIZE;
  const int warpId = fid / WARP_SIZE;
  const int blockSize = blockDim.x * blockDim.y;

  float Q_i = Q[rid * hf + hfid];

  for (int j = 0; j < num_neighbor; j++)
  {
    float weight_partial = 0;

    int cid = indices[lb + j];
    weight_partial = Q_i * K[cid * hf + hfid];

    __syncthreads();
    weight_partial = warpReduceSum(weight_partial, blockSize);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
    if (fid == 0)
    {
      attn_edge[lb + j] = weight_partial * val[lb + j];
    }
  }
  __syncthreads();
}

__global__ void softMax_SPMM(const int m, const int nnz, const int h, const int f,
                             const int *indptr, const int *indices, const float *val,
                             const float *V, const float *attn_edge,
                             float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *neigh_nodes_weight = smem;
  float weightMax = -1e38;
  const int hf = h * f;
  const int hfid = hid * f + fid;

  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    if (fid == 0)
    {
      neigh_nodes_weight[j] = attn_edge[lb + j];
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = indices[lb + j];
    float weight = neigh_nodes_weight[j];
    attn_val = exp(weight - weightMax) / expAll;
    acc += attn_val * V[cid * hf + hfid];
    __syncthreads();
  }

  out_feat[rid * hf + hfid] = acc;
}

template <typename DType, int blockSize, int LOG_BLOCK_SIZE>
__global__ void fused_forward_kernel_subgraph(const int h, const int f,
                                              const int *node_num_ptr, const int *indptr, const int *indices, const DType *val,
                                              const DType *Q, const DType *K, const DType *V,
                                              DType *out_feat)
{
  // grid: 4096*h  block: 32 * 8 each tb processes one graph
  // blockSize = blockDim.y
  const int gid = blockIdx.x;   // index of subgraph
  const int hid = blockIdx.y;   // index of head
  const int tidx = threadIdx.x; // index of WARP
  const int tidy = threadIdx.y;
  const int node_lb = node_num_ptr[gid];
  const int node_hb = node_num_ptr[gid + 1]; // Offset of nodes in the subgraph on the full graph
  const int num_nodes = node_hb - node_lb;   // num of nodes in this subgraph

  const int hf = h * f;
  extern __shared__ DType smem[];
  DType *K_SMEM = smem;
  DType *V_SMEM = (DType *)&K_SMEM[num_nodes * hf];
  DType *neigh_nodes_weight = (DType *)&V_SMEM[num_nodes * hf];

  int loops_node = (num_nodes + blockSize - 1) / blockSize;
  int loops_feat = (f + WARP_SIZE - 1) / WARP_SIZE;
  // Put the K and V into smem
  for (int j = 0; j < loops_node; j++)
  {
    int curr_node = j * blockSize + tidy;
    if (curr_node + node_lb < node_hb)
    {

      for (int i = 0; i < loops_feat; i++)
      {
        int curr_feat = tidx + (i << 5);
        if (curr_feat < f)
        {
          K_SMEM[curr_node * hf + hid * f + curr_feat] = K[(node_lb + curr_node) * hf + hid * f + curr_feat];
          V_SMEM[curr_node * hf + hid * f + curr_feat] = V[(node_lb + curr_node) * hf + hid * f + curr_feat];
        }
      }
    }
  }

  __syncthreads();

  for (int j = 0; j < loops_node; j++)
  {
    int curr_node = tidy + j * blockSize;
    if (curr_node + node_lb < node_hb)
    {
      DType weightMax = -1e38;
      const int lb = indptr[node_lb + curr_node]; // row rid elements
      const int hb = indptr[node_lb + curr_node + 1];
      const int num_neighbor = hb - lb;
      for (int k = 0; k < num_neighbor; k++)
      {
        int cid = indices[lb + k] - node_lb;
        DType weight_partial = 0;
        for (int i = 0; i < loops_feat; i++)
        {
          int curr_feat = tidx + (i << 5);
          if (curr_feat < f)
          {
            DType Q_i = Q[(node_lb + curr_node) * hf + hid * f + curr_feat];
            weight_partial += Q_i * K_SMEM[cid * hf + hid * f + curr_feat];
          }
        }
        __syncthreads();
        // weight_partial = warpReduceSum(weight_partial, WARP_SIZE);
        for (int stride = 16; stride > 0; stride >>= 1)
        {
          weight_partial += __shfl_xor_sync(0xffffffff, weight_partial, stride, 32);
        }
        __syncwarp();
        if (tidx == 0)
        {
          neigh_nodes_weight[tidy + (k << LOG_BLOCK_SIZE)] = weight_partial * val[lb + k];
        }
        __syncthreads();
        weightMax = MAX(weight_partial, weightMax);
      }
      __syncthreads();

      int loop_WARP_neigh = (num_neighbor + WARP_SIZE - 1) / WARP_SIZE;
      DType expAll = 0;
      for (int k = 0; k < loop_WARP_neigh; k++)
      {
        DType exptmp = 0;
        int pid = tidx + (k << 5);
        if (pid < num_neighbor)
        {
          DType weight = neigh_nodes_weight[tidy + (pid << LOG_BLOCK_SIZE)];
          exptmp = exp(weight - weightMax);
        }
        __syncwarp();
        for (int stride = 16; stride > 0; stride >>= 1)
        {
          exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
        }
        __syncwarp();
        expAll += exptmp;
      }
      __syncthreads();

      // compute the output
      for (int i = 0; i < loops_feat; i++)
      {
        DType acc = 0;
        DType attn_val;
        int curr_feat = tidx + (i << 5);
        if (curr_feat < f)
        {
          for (int k = 0; k < num_neighbor; k++)
          {
            int cid = indices[lb + k] - node_lb;
            DType weight = neigh_nodes_weight[tidy + (k << LOG_BLOCK_SIZE)];
            attn_val = exp(weight - weightMax) / expAll;
            acc += attn_val * V_SMEM[cid * hf + hid * f + curr_feat];
          }
        }
        __syncthreads();
        out_feat[(node_lb + curr_node) * hf + hid * f + curr_feat] = acc;
      }
    }
    __syncthreads();
  }
}

__global__ void fused_forward_kernel_mul32(const int m, const int nnz, const int h, const int f,
                                           const int *indptr, const int *indices, const float *val,
                                           const float *Q, const float *K, const float *V,
                                           float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *neigh_nodes_weight = smem;
  float weightMax = -1e38;
  static __shared__ float warpLevelSums[WARP_SIZE];
  const int hf = h * f;
  const int hfid = hid * f + fid;
  const int laneId = fid % WARP_SIZE;
  const int warpId = fid / WARP_SIZE;
  float Q_i = Q[rid * hf + hfid];

  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    float weight_partial = 0;

    int cid = indices[lb + j];
    weight_partial = Q_i * K[cid * hf + hfid];

    __syncthreads();
    weight_partial = warpReduceSum(weight_partial, f);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < f / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, f / WARP_SIZE);
    if (fid == 0)
    {
      neigh_nodes_weight[j] = weight_partial * val[lb + j];
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  __syncthreads();

  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = indices[lb + j];
    float weight = neigh_nodes_weight[j];
    attn_val = exp(weight - weightMax) / expAll;
    acc += attn_val * V[cid * hf + hfid];
    __syncthreads();
  }

  out_feat[rid * hf + hfid] = acc;
}

__global__ void fused_forward_kernel(const int m, const int nnz, const int h, const int f,
                                     const int *indptr, const int *indices, const float *val,
                                     const float *Q, const float *K, const float *V,
                                     float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = indptr[rid]; // row rid elements
  const int hb = indptr[rid + 1];

  const int threads_x = blockDim.x; // 32
  const int threads_y = blockDim.y; // f/32
  const int blockSize = threads_x * threads_y;
  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *curr_node_feature = smem;
  float *neigh_nodes_weight = (float *)&curr_node_feature[f];
  float weightMax = -1e38;
  // init the shared memory
  float Q_i = 0;
  if (fid < f)
  {
    Q_i = Q[rid * h * f + hid * f + fid];
  }

  // compute the attention weight
  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    float weight_partial = 0;
    if (fid < f)
    {
      int cid = indices[lb + j];
      weight_partial = Q_i * K[cid * h * f + hid * f + fid];
    }
    __syncthreads();
    static __shared__ float warpLevelSums[WARP_SIZE];
    const int laneId = fid % WARP_SIZE;
    const int warpId = fid / WARP_SIZE;
    weight_partial = warpReduceSum(weight_partial, blockSize);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
    if (fid == 0)
    {
      neigh_nodes_weight[j] = weight_partial;
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  __syncthreads();

  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = indices[lb + j];
    if (fid < f)
    {
      float weight = neigh_nodes_weight[j];
      attn_val = exp(weight - weightMax) / expAll;
      acc += attn_val * V[cid * h * f + hid * f + fid];
    }
    __syncthreads();
  }
  if (fid < f)
    out_feat[rid * h * f + hid * f + fid] = acc;
}

__global__ void fused_forward_ell_kernel(const int m, const int nnz, const int h, const int f,
                                         const int *indptr, const int *indices,
                                         const int *row_index, const int *rows_per_tb, const float *val,
                                         const float *Q, const float *K, const float *V,
                                         float *out_feat)
{
  const int bid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim
  const int lrow = rows_per_tb[bid];              // row rid elements
  const int hrow = rows_per_tb[bid + 1];
  const int num_rows = hrow - lrow;
  extern __shared__ float smem[];
  float *curr_node_feature = smem;
  float *neigh_nodes_weight = (float *)&curr_node_feature[f];

  const int threads_x = blockDim.x; // 32
  const int threads_y = blockDim.y; // f/32
  const int blockSize = threads_x * threads_y;
  float weightMax = -1e38;

  for (int row = 0; row < num_rows; row++)
  {
    int rid = row_index[lrow + row];
    const int lb = indptr[rid]; // row rid elements
    const int hb = indptr[rid + 1];

    const int num_neighbor = hb - lb;

    // init the shared memory
    if (fid < f)
    {
      curr_node_feature[fid] = Q[rid * h * f + hid * f + fid];
    }

    // compute the attention weight
    for (int j = 0; j < num_neighbor; j++)
    {
      float weight = 0;
      float weight_partial = 0;
      if (fid < f)
      {
        int cid = indices[lb + j];
        weight_partial = curr_node_feature[fid] * K[cid * h * f + hid * f + fid];
      }
      __syncthreads();
      static __shared__ float warpLevelSums[WARP_SIZE];
      const int laneId = fid % WARP_SIZE;
      const int warpId = fid / WARP_SIZE;
      weight_partial = warpReduceSum(weight_partial, blockSize);
      if (laneId == 0)
        warpLevelSums[warpId] = weight_partial;
      __syncthreads();
      weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
      if (warpId == 0)
        weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
      if (fid == 0)
      {
        neigh_nodes_weight[j] = weight_partial;
      }
      __syncthreads();
      weight = neigh_nodes_weight[j];
      weightMax = MAX(weight, weightMax);
    }
    __syncthreads();

    // compute the sum of exp
    int loop = (num_neighbor + 31) / 32;
    float expAll = 0;
    for (int j = 0; j < loop; j++)
    {
      int pid = threadIdx.x + (j << 5); // node need to process in loop j
      float exptmp = 0;
      if (pid < num_neighbor)
      {
        float weight = neigh_nodes_weight[pid];
        exptmp = exp(weight - weightMax);
      }
      __syncwarp();
      for (int stride = 16; stride > 0; stride >>= 1)
      {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }
    __syncthreads();

    // compute the output
    float acc = 0;
    for (int j = 0; j < num_neighbor; j++)
    {
      float attn_val;
      int cid = indices[lb + j];
      if (fid < f)
      {
        float weight = neigh_nodes_weight[j];
        attn_val = exp(weight - weightMax) / expAll;
        acc += attn_val * V[cid * h * f + hid * f + fid];
      }
      __syncthreads();
    }
    if (fid < f)
      out_feat[rid * h * f + hid * f + fid] = acc;
    __syncthreads();
  }
}

void gf_forward(int m, int nnz, int h, int f,
                const int *indptr, const int *indices, const float *val,
                const float *Q, const float *K, const float *V,
                float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  const dim3 nblks(m, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (fused_forward_kernel),
      nblks, nthrs, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
      Q, K, V, out_feat);
  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // float elapsedTime;
  // hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("Time of fused kernel: %f \n", elapsedTime);
}

void gf_forward_subgraph(int num_subgraph, int h, int f, const int *nodes_subgraph,
                         const int *indptr, const int *indices, const float *val,
                         const float *Q, const float *K, const float *V,
                         float *out_feat)
{
  const int BLOCK_SIZE = atoi(getenv("BLOCK_SIZE"));

  const int ntx = 32;         // on feature dimension
  const int nty = BLOCK_SIZE; // on out dimension
  const int nbx = num_subgraph;
  const int nby = h;
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);
  switch (BLOCK_SIZE)
  {
  case 8:
    hipFuncSetAttribute(reinterpret_cast<const void*>(fused_forward_kernel_subgraph<float), 8, 3>, hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_subgraph<float, 8, 3>),
        nblks, nthrs, 1024 * 64, h, f, nodes_subgraph, indptr, indices, val,
        Q, K, V, out_feat);
    break;
  case 32:
    hipFuncSetAttribute(reinterpret_cast<const void*>(fused_forward_kernel_subgraph<float), 32, 5>, hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_subgraph<float, 32, 5>),
        nblks, nthrs, 1024 * 64, h, f, nodes_subgraph, indptr, indices, val,
        Q, K, V, out_feat);
    break;
  default:
    throw "not supported BLOCKSIZE!";
  }
}

void gf_forward_nofuse(int m, int nnz, int h, int f,
                       const int *indptr, const int *indices, const int *rows, const float *val,
                       const float *Q, const float *K, const float *V,
                       float *attn_edge, float *out_feat)
{
  const int ntx = 32; // on feature dimension
  const int nty = 8;  // on out dimension
  const int nbx = (nnz + nty - 1) / nty;
  const int nby = FindNumBlocks<'y'>(h);
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);

  CUDA_KERNEL_CALL(
      (sddmmCooKernel<float>),
      nblks, nthrs, 0, f * h, f * h, h, nnz, f, rows, indices, val,
      Q, K, attn_edge);

  const dim3 nblks2(m, h, 1);
  const dim3 nthrs2(32, (f + 31) / 32, 1);
  // CUDA_KERNEL_CALL(
  //     (sddmmCsrKernel),
  //     nblks2, nthrs2, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
  //     Q, K, attn_edge);

  // const dim3 nblks2(m, h, 1);
  // const dim3 nthrs2(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (softMax_SPMM),
      nblks2, nthrs2, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
      V, attn_edge, out_feat);
}

void gf_forward_multiple32(int m, int nnz, int h, int f,
                           const int *indptr, const int *indices, const float *val,
                           const float *Q, const float *K, const float *V,
                           float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);
  const dim3 nblks(m, h, 1);
  const dim3 nthrs(32, f / 32, 1);
  CUDA_KERNEL_CALL(
      (fused_forward_kernel_mul32),
      nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, indptr, indices, val,
      Q, K, V, out_feat);
}

void gf_ell_forward(int m, int nnz, int h, int f, int num_tb,
                    const int *indptr, const int *indices,
                    const int *row_index, const int *rows_per_tb, const float *val,
                    const float *Q, const float *K, const float *V,
                    float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  const dim3 nblks(num_tb, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (fused_forward_ell_kernel),
      nblks, nthrs, (f + 512) * sizeof(float), m, nnz, h, f, indptr, indices, row_index, rows_per_tb, val,
      Q, K, V, out_feat);
  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // float elapsedTime;
  // hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("Time of fused kernel: %f \n", elapsedTime);
}

std::vector<torch::Tensor>
gf_forward_cuda(torch::Tensor indptr,
                torch::Tensor indices,
                torch::Tensor val, torch::Tensor Q,
                torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = indptr.size(0) - 1; // num of nodes
  const auto nnz = indices.size(0);  // num of edges
  const auto h = Q.size(1);          // num of heads
  const auto f = Q.size(2);          // num of feats
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);

  // check whether f is multiples of 32
  if (isMul32(f))
  {
    gf_forward_multiple32(m, nnz, h, f,
                          indptr.data_ptr<int>(), indices.data_ptr<int>(), val.data_ptr<float>(),
                          Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                          out_feat.data_ptr<float>());
  }
  else
  {
    gf_forward(m, nnz, h, f,
               indptr.data_ptr<int>(), indices.data_ptr<int>(), val.data_ptr<float>(),
               Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
               out_feat.data_ptr<float>());
  }
  return {out_feat};
}

std::vector<torch::Tensor>
gf_subgraph_forward_cuda(torch::Tensor nodes_subgraph,
                         torch::Tensor indptr,
                         torch::Tensor indices,
                         torch::Tensor val, torch::Tensor Q,
                         torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto num_subgraph = nodes_subgraph.size(0) - 1;
  const auto m = indptr.size(0) - 1; // num of nodes
  const auto h = Q.size(1);          // num of heads
  const auto f = Q.size(2);          // num of feats
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  gf_forward_subgraph(num_subgraph, h, f, nodes_subgraph.data_ptr<int>(),
                      indptr.data_ptr<int>(), indices.data_ptr<int>(), val.data_ptr<float>(),
                      Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                      out_feat.data_ptr<float>());

  return {out_feat};
}

std::vector<torch::Tensor>
gf_hyper_forward_cuda(torch::Tensor indptr,
                      torch::Tensor indices, torch::Tensor rows,
                      torch::Tensor val, torch::Tensor Q,
                      torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = indptr.size(0) - 1; // num of nodes
  const auto nnz = indices.size(0);  // num of edges
  const auto h = Q.size(1);          // num of heads
  const auto f = Q.size(2);          // num of feats
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  auto attn_edge = torch::zeros({nnz * h}, options);
  gf_forward_nofuse(m, nnz, h, f,
                    indptr.data_ptr<int>(), indices.data_ptr<int>(), rows.data_ptr<int>(), val.data_ptr<float>(),
                    Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                    attn_edge.data_ptr<float>(), out_feat.data_ptr<float>());

  return {out_feat};
}

std::vector<torch::Tensor>
gf_ell_forward_cuda(torch::Tensor indptr,
                    torch::Tensor indices,
                    torch::Tensor row_index,
                    torch::Tensor rows_per_tb,
                    torch::Tensor val, torch::Tensor Q,
                    torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = indptr.size(0) - 1;           // num of nodes
  const auto nnz = indices.size(0);            // num of edges
  const auto h = Q.size(1);                    // num of heads
  const auto f = Q.size(2);                    // num of feats
  const auto num_tb = rows_per_tb.size(0) - 1; // num of thread blocks
  auto devid = indptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  gf_ell_forward(m, nnz, h, f, num_tb,
                 indptr.data_ptr<int>(), indices.data_ptr<int>(),
                 row_index.data_ptr<int>(), rows_per_tb.data_ptr<int>(), val.data_ptr<float>(),
                 Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                 out_feat.data_ptr<float>());
  return {out_feat};
}