#include "hip/hip_runtime.h"
#include "../util/computeUtil.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>
#include <unistd.h>
#include <stdio.h>

using namespace std;

const int WARP_SIZE = 32;

extern "C" bool isPow2(unsigned int x) { return ((x & (x - 1)) == 0); }

#define CUDA_CALL(func)                                      \
  {                                                          \
    hipError_t e = (func);                                  \
    CHECK(e == hipSuccess || e == hipErrorDeinitialized) \
        << "CUDA: " << hipGetErrorString(e);                \
  }

#define CUSPARSE_CALL(func)                                         \
  {                                                                 \
    hipsparseStatus_t e = (func);                                    \
    CHECK(e == HIPSPARSE_STATUS_SUCCESS) << "CUSPARSE ERROR: " << e; \
  }

#define CUDA_KERNEL_CALL(kernel, nblks, nthrs, shmem, ...)          \
  {                                                                 \
    {                                                               \
      (kernel)<<<(nblks), (nthrs), (shmem)>>>(__VA_ARGS__);         \
      hipError_t e = hipGetLastError();                           \
      CHECK(e == hipSuccess || e == hipErrorDeinitialized)      \
          << "CUDA kernel launch error: " << hipGetErrorString(e); \
    }                                                               \
  }

__device__ __forceinline__ float warpReduceSum(float sum, int blockSize)
{
  if (blockSize >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16);
  if (blockSize >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8);
  if (blockSize >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4);
  if (blockSize >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2);
  if (blockSize >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1);
  return sum;
}

// __device__ __forceinline__ float warpReduceSum(unsigned int mask, float mySum)
// {
//   for (int offset = warpSize / 2; offset > 0; offset /= 2)
//   {
//     mySum += __shfl_down_sync(mask, mySum, offset);
//   }
//   return mySum;
// }


template <unsigned int blockSize>
__global__ void fused_forward_kernel_pow2(const int m, const int nnz, const int h, const int f,
                                          const int *row_ptr, const int *col_ind, const float *val,
                                          const float *Q, const float *K, const float *V,
                                          float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = row_ptr[rid]; // row rid elements
  const int hb = row_ptr[rid + 1];

  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *neigh_nodes_weight = smem;
  float weightMax = -1e38;
  static __shared__ float warpLevelSums[WARP_SIZE];
  const int hf = h * f;
  const int hfid = hid * f + fid;
  const int laneId = fid % WARP_SIZE;
  const int warpId = fid / WARP_SIZE;
  float Q_i = Q[rid * hf + hfid];

  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    float weight_partial = 0;

    int cid = col_ind[lb + j];
    weight_partial = Q_i * K[cid * hf + hfid];

    __syncthreads();
    weight_partial = warpReduceSum(weight_partial, blockSize);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
    if (fid == 0)
    {
      neigh_nodes_weight[j] = weight_partial * val[lb + j];
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  __syncthreads();

  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = col_ind[lb + j];
    float weight = neigh_nodes_weight[j];
    attn_val = exp(weight - weightMax) / expAll;
    acc += attn_val * V[cid * hf + hfid];
    __syncthreads();
  }

  out_feat[rid * hf + hfid] = acc;
}

__global__ void fused_forward_kernel(const int m, const int nnz, const int h, const int f,
                                     const int *row_ptr, const int *col_ind, const float *val,
                                     const float *Q, const float *K, const float *V,
                                     float *out_feat)
{
  const int rid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim

  const int lb = row_ptr[rid]; // row rid elements
  const int hb = row_ptr[rid + 1];

  const int threads_x = blockDim.x; // 32
  const int threads_y = blockDim.y; // f/32
  const int blockSize = threads_x * threads_y;
  const int num_neighbor = hb - lb;
  extern __shared__ float smem[];
  float *curr_node_feature = smem;
  float *neigh_nodes_weight = (float *)&curr_node_feature[f];
  float weightMax = -1e38;
  // init the shared memory
  float Q_i = 0;
  if (fid < f)
  {
    Q_i = Q[rid * h * f + hid * f + fid];
    // curr_node_feature[fid] = Q[rid * h * f + hid * f + fid];
  }

  // compute the attention weight
  for (int j = 0; j < num_neighbor; j++)
  {
    float weight = 0;
    float weight_partial = 0;
    if (fid < f)
    {
      int cid = col_ind[lb + j];
      weight_partial = Q_i * K[cid * h * f + hid * f + fid];
    }
    __syncthreads();
    static __shared__ float warpLevelSums[WARP_SIZE];
    const int laneId = fid % WARP_SIZE;
    const int warpId = fid / WARP_SIZE;
    weight_partial = warpReduceSum(weight_partial, blockSize);
    if (laneId == 0)
      warpLevelSums[warpId] = weight_partial;
    __syncthreads();
    weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
    if (warpId == 0)
      weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
    if (fid == 0)
    {
      neigh_nodes_weight[j] = weight_partial;
    }
    __syncthreads();
    weight = neigh_nodes_weight[j];
    weightMax = MAX(weight, weightMax);
  }
  __syncthreads();

  // compute the sum of exp
  int loop = (num_neighbor + 31) / 32;
  float expAll = 0;
  for (int j = 0; j < loop; j++)
  {
    int pid = threadIdx.x + (j << 5); // node need to process in loop j
    float exptmp = 0;
    if (pid < num_neighbor)
    {
      float weight = neigh_nodes_weight[pid];
      exptmp = exp(weight - weightMax);
    }
    __syncwarp();
    for (int stride = 16; stride > 0; stride >>= 1)
    {
      exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
    }
    __syncwarp();
    expAll += exptmp;
  }
  __syncthreads();

  // compute the output
  float acc = 0;
  for (int j = 0; j < num_neighbor; j++)
  {
    float attn_val;
    int cid = col_ind[lb + j];
    if (fid < f)
    {
      float weight = neigh_nodes_weight[j];
      attn_val = exp(weight - weightMax) / expAll;
      acc += attn_val * V[cid * h * f + hid * f + fid];
    }
    __syncthreads();
  }
  if (fid < f)
    out_feat[rid * h * f + hid * f + fid] = acc;
}

__global__ void fused_forward_ell_kernel(const int m, const int nnz, const int h, const int f,
                                         const int *row_ptr, const int *col_ind,
                                         const int *row_index, const int *rows_per_tb, const float *val,
                                         const float *Q, const float *K, const float *V,
                                         float *out_feat)
{
  const int bid = blockIdx.x;                     // loop over row of adj matrix
  const int hid = blockIdx.y;                     // loop over heads
  const int fid = threadIdx.y * 32 + threadIdx.x; // loop over feature dim
  const int lrow = rows_per_tb[bid];              // row rid elements
  const int hrow = rows_per_tb[bid + 1];
  const int num_rows = hrow - lrow;
  extern __shared__ float smem[];
  float *curr_node_feature = smem;
  float *neigh_nodes_weight = (float *)&curr_node_feature[f];

  const int threads_x = blockDim.x; // 32
  const int threads_y = blockDim.y; // f/32
  const int blockSize = threads_x * threads_y;
  float weightMax = -1e38;

  for (int row = 0; row < num_rows; row++)
  {
    int rid = row_index[lrow + row];
    const int lb = row_ptr[rid]; // row rid elements
    const int hb = row_ptr[rid + 1];

    const int num_neighbor = hb - lb;

    // init the shared memory
    if (fid < f)
    {
      curr_node_feature[fid] = Q[rid * h * f + hid * f + fid];
    }

    // compute the attention weight
    for (int j = 0; j < num_neighbor; j++)
    {
      float weight = 0;
      float weight_partial = 0;
      if (fid < f)
      {
        int cid = col_ind[lb + j];
        weight_partial = curr_node_feature[fid] * K[cid * h * f + hid * f + fid];
      }
      __syncthreads();
      static __shared__ float warpLevelSums[WARP_SIZE];
      const int laneId = fid % WARP_SIZE;
      const int warpId = fid / WARP_SIZE;
      weight_partial = warpReduceSum(weight_partial, blockSize);
      if (laneId == 0)
        warpLevelSums[warpId] = weight_partial;
      __syncthreads();
      weight_partial = (fid < blockSize / WARP_SIZE) ? warpLevelSums[laneId] : 0;
      if (warpId == 0)
        weight_partial = warpReduceSum(weight_partial, blockSize / WARP_SIZE);
      if (fid == 0)
      {
        neigh_nodes_weight[j] = weight_partial;
      }
      __syncthreads();
      weight = neigh_nodes_weight[j];
      weightMax = MAX(weight, weightMax);
    }
    __syncthreads();

    // compute the sum of exp
    int loop = (num_neighbor + 31) / 32;
    float expAll = 0;
    for (int j = 0; j < loop; j++)
    {
      int pid = threadIdx.x + (j << 5); // node need to process in loop j
      float exptmp = 0;
      if (pid < num_neighbor)
      {
        float weight = neigh_nodes_weight[pid];
        exptmp = exp(weight - weightMax);
      }
      __syncwarp();
      for (int stride = 16; stride > 0; stride >>= 1)
      {
        exptmp += __shfl_xor_sync(0xffffffff, exptmp, stride, 32);
      }
      __syncwarp();
      expAll += exptmp;
    }
    __syncthreads();

    // compute the output
    float acc = 0;
    for (int j = 0; j < num_neighbor; j++)
    {
      float attn_val;
      int cid = col_ind[lb + j];
      if (fid < f)
      {
        float weight = neigh_nodes_weight[j];
        attn_val = exp(weight - weightMax) / expAll;
        acc += attn_val * V[cid * h * f + hid * f + fid];
      }
      __syncthreads();
    }
    if (fid < f)
      out_feat[rid * h * f + hid * f + fid] = acc;
    __syncthreads();
  }
}

void gf_forward(int m, int nnz, int h, int f,
                const int *row_ptr, const int *col_ind, const float *val,
                const float *Q, const float *K, const float *V,
                float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  const dim3 nblks(m, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (fused_forward_kernel),
      nblks, nthrs, (f + 512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
      Q, K, V, out_feat);
  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // float elapsedTime;
  // hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("Time of fused kernel: %f \n", elapsedTime);
}

void gf_forward_pow2(int m, int nnz, int h, int f,
                     const int *row_ptr, const int *col_ind, const float *val,
                     const float *Q, const float *K, const float *V,
                     float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);
  const dim3 nblks(m, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  switch (f)
  {
  case 4096:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<4096>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;
  case 2048:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<2048>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;
  case 1024:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<1024>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;
  case 512:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<512>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;

  case 256:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<256>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;

  case 128:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<128>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;

  case 64:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<64>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;

  case 32:
    CUDA_KERNEL_CALL(
        (fused_forward_kernel_pow2<32>),
        nblks, nthrs, (512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, val,
        Q, K, V, out_feat);
    break;
  }

  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // float elapsedTime;
  // hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("Time of fused kernel: %f \n", elapsedTime);
}

void gf_ell_forward(int m, int nnz, int h, int f, int num_tb,
                    const int *row_ptr, const int *col_ind,
                    const int *row_index, const int *rows_per_tb, const float *val,
                    const float *Q, const float *K, const float *V,
                    float *out_feat)
{
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  const dim3 nblks(num_tb, h, 1);
  const dim3 nthrs(32, (f + 31) / 32, 1);
  CUDA_KERNEL_CALL(
      (fused_forward_ell_kernel),
      nblks, nthrs, (f + 512) * sizeof(float), m, nnz, h, f, row_ptr, col_ind, row_index, rows_per_tb, val,
      Q, K, V, out_feat);
  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // float elapsedTime;
  // hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("Time of fused kernel: %f \n", elapsedTime);
}

std::vector<torch::Tensor>
gf_forward_cuda(torch::Tensor row_ptr,
                torch::Tensor col_ind,
                torch::Tensor val, torch::Tensor Q,
                torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = row_ptr.size(0) - 1; // num of nodes
  const auto nnz = col_ind.size(0);   // num of edges
  const auto h = Q.size(1);           // num of heads
  const auto f = Q.size(2);           // num of feats
  auto devid = row_ptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  if (isPow2(f))
  {
    gf_forward_pow2(m, nnz, h, f,
                    row_ptr.data_ptr<int>(), col_ind.data_ptr<int>(), val.data_ptr<float>(),
                    Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                    out_feat.data_ptr<float>());
  }
  else
  {
    gf_forward(m, nnz, h, f,
               row_ptr.data_ptr<int>(), col_ind.data_ptr<int>(), val.data_ptr<float>(),
               Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
               out_feat.data_ptr<float>());
  }
  return {out_feat};
}

std::vector<torch::Tensor>
gf_ell_forward_cuda(torch::Tensor row_ptr,
                    torch::Tensor col_ind,
                    torch::Tensor row_index,
                    torch::Tensor rows_per_tb,
                    torch::Tensor val, torch::Tensor Q,
                    torch::Tensor K, torch::Tensor V)
{
  // Q: torch.Size([6248, 10, 8])
  const auto m = row_ptr.size(0) - 1;          // num of nodes
  const auto nnz = col_ind.size(0);            // num of edges
  const auto h = Q.size(1);                    // num of heads
  const auto f = Q.size(2);                    // num of feats
  const auto num_tb = rows_per_tb.size(0) - 1; // num of thread blocks
  auto devid = row_ptr.device().index();
  auto options =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto out_feat = torch::zeros({m, h, f}, options);
  gf_ell_forward(m, nnz, h, f, num_tb,
                 row_ptr.data_ptr<int>(), col_ind.data_ptr<int>(),
                 row_index.data_ptr<int>(), rows_per_tb.data_ptr<int>(), val.data_ptr<float>(),
                 Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
                 out_feat.data_ptr<float>());
  return {out_feat};
}